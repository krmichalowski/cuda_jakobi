#include "hip/hip_runtime.h"
#include "func.h"
#include<iostream>

void plan_distribution(int gpu_size, int n_threads, int* per_thread, int* red_n_threads, int* last_filled)
{
    /*calcuate some cuantities so that they wont have to be cacluated by every gpu thread*/
    per_thread[0] = (gpu_size - 2)/n_threads;
    red_n_threads[0] = (gpu_size - 2) - per_thread[0] * n_threads;
    last_filled[0] = (n_threads - 1) * per_thread[0] + per_thread[0] + 1;
}

__global__ void set_boundry(double* gpu_x, double* rhs, double* gpu_x_new, double x0, double xn, int gpu_size, int size)
{
    /*set boundry values int gpu memory space, they are not changing while calcuating, something
    resembling dirichlet BC*/
    gpu_x[0] = x0;
    gpu_x[gpu_size - 1] = xn;

    gpu_x_new[0] = x0;
    gpu_x_new[gpu_size - 1] = xn;

    rhs[0] = x0;
    rhs[size - 1] = xn;
}

__global__ void set_single_boundry(double* gpu_solution, int size, double x0, double xn)
{
    /*set boundry values int gpu memory space, they are not changing while calcuating, something
    resembling dirichlet BC*/
    gpu_solution[0] = x0;
    gpu_solution[size - 1] = xn;
}

__global__ void initial_guess_begin(double* gpu_x, double guess, int per_thread)
{
    /*set initial guess except for the boundry, (gpu size - 4) is not divisible by number of threds
    due to the possibly large number of threads (and the fact that they are slover then cpu threads)
    i didnt want to put a lot of extra work, for eg, on the last thread, thats why filling with initial
    guess is split into two functions. This function is filling most of the array by simply equally
    distributing the work, per_thread = (gpu size - 4)/2 but rounded down to an int.*/

    /*in this case all this could have been avoided by just filling the whole array with initial guess
    and than overwirting the boundry, but for other cases this strategy might be useful*/
    int id = threadIdx.x;

    int i;
    int start = 1 + id * per_thread;
    for(i=start;i<start+per_thread;i++)
    {
        gpu_x[i] = guess;
    }
}

__global__ void initial_guess_finish(double* gpu_x, double guess, int per_thread, int last_filled)
{
    /*this function is called with approprietly reduced number of threads, every thread gets to set one
    value, this function finishes filling after initial_guess_begin (reduced n_threads is calculated by
    plan_distribution function)*/
    int id = threadIdx.x;
    gpu_x[id + last_filled] = guess;
}

__global__ void jacobi_step(double* gpu_x, double* gpu_x_new, double* rhs, int block_size)
{
    /*single jacobi iteration, padded nodes in a block are not calculated, they are just there to enable
    the independend calculation for every block*/
    int id = threadIdx.x;
    int i;
    int offset = 2 * id;
    for(i=1+block_size*id;i<(1+id)*block_size-1;i++)
    {
        gpu_x_new[i] = (rhs[i - offset] - gpu_x[i - 1] - gpu_x[i + 1])/4;
    }
}

__global__ void rhs_fill(double* rhs, int size, int per_thread)
{
    int id = threadIdx.x;
    int i;
    for(i=1+id*per_thread;i<1+(id+1)*per_thread;i++)
    {
        rhs[i] = 5 - 4 * ((double)i/(size-1));
    }
}

__global__ void update_padded(double* gpu_new, int block_size)
{
    /*this updates the padded nodes, every thread updates two neigbouring padded nodes with values
    calcuated befour in jacobi_step. This just copies the values from appropriate place in the same
    array.*/
    int id = threadIdx.x;
    int index = (id + 1) * block_size - 1;

    gpu_new[index] = gpu_new[index + 2];
    gpu_new[index + 1] = gpu_new[index - 1];
}

void test_solution(double* rhs, double* x, int size)
{
    int i;
    for(i=1;i<size-1;i++)
    {
        printf("%lf   %d\n", rhs[i] - 4*x[i] - 1*x[i + 1] - 1*x[i - 1], i);
        //printf("%lf\n", x[i]);
    }
}

__global__ void flatten_solution(double* gpu_x, double* gpu_x_flatten, int size, int block_size)
{
    /*go from padded vector to vector without padding, due to padding not being important in displaying
    or testing solution*/
    int id = threadIdx.x;

    int i;
    int offset = 2 * id;
    for(i=id*block_size+1;i<id*block_size+1+block_size-2;i++)
    {
        gpu_x_flatten[i - offset] = gpu_x[i];
    }
}

void jacobi_solve(int n_iter, double* gpu_x, double* gpu_x_new, double* gpu_rhs, int block_size, double* solution, int size, int n_threads)
{
    /*wrapper function for calling other functions in appropriate order*/
    int i = 0;
    double* gpu_solution;
    hipMalloc(&gpu_solution, size * sizeof(double));
    for(i=0;i<n_iter;i++)
    {
        jacobi_step<<<1,n_threads>>>(gpu_x, gpu_x_new, gpu_rhs, block_size);
        update_padded<<<1,n_threads-1>>>(gpu_x_new, block_size);
        double* tmp = gpu_x;
        gpu_x = gpu_x_new;
        gpu_x_new = tmp;
    }

    set_single_boundry<<<1,1>>>(gpu_solution, size, 5.0, 1.0);

    flatten_solution<<<1,n_threads>>>(gpu_x, gpu_solution, size, block_size);

    hipMemcpy(solution, gpu_solution, size * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(gpu_solution);
}


