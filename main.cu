#include <stdio.h>
#include "func.h"


int main()
{
    int size = 514;
    int n_threads = 256;

    int block_size = ((size - 2)/n_threads) + 2;
    int gpu_size = size + 2 * (n_threads) - 2;

    double* solution = (double*)malloc(size * sizeof(double));
    double* rhs = (double*)malloc(size * sizeof(double));

    double* gpu_x;
    double* gpu_x_new;
    double* gpu_solution;
    double* gpu_rhs;
    hipMalloc(&gpu_x, gpu_size * sizeof(double));
    hipMalloc(&gpu_x_new, gpu_size * sizeof(double));
    hipMalloc(&gpu_rhs, size * sizeof(double));
    hipMalloc(&gpu_solution, size * sizeof(double));

    double x0 = 5.0;
    double xn = 1.0;
    set_boundry<<<1,1>>>(gpu_x, gpu_rhs, gpu_x_new, gpu_solution, x0, xn, gpu_size, size);
    rhs_fill<<<1,n_threads>>>(gpu_rhs, size, (size - 2)/n_threads);

    double guess = 2.0;
    int per_thread;
    int red_n_threads;
    int last_filled;
    plan_distribution(gpu_size, n_threads, &per_thread, &red_n_threads, &last_filled);
    initial_guess_begin<<<1,n_threads>>>(gpu_x, guess, per_thread);
    initial_guess_finish<<<1,red_n_threads>>>(gpu_x, guess, per_thread, last_filled);

    jacobi_step<<<1,n_threads>>>(gpu_x, gpu_x_new, rhs, block_size);

    flatten_solution<<<1,n_threads>>>(gpu_x_new, gpu_solution, size, block_size);

    hipMemcpy(solution, gpu_solution, size * sizeof(double), hipMemcpyDeviceToHost);

    hipMemcpy(rhs, gpu_rhs, size * sizeof(double), hipMemcpyDeviceToHost);
    test_solution(rhs, rhs, size);
}